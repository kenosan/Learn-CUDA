
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int a, int b, int *c) {
	*c = a + b;
}


int main(void) {
	int c, *dev_c;

	// Allocate memory on the device
	hipMalloc( (void**)&dev_c, sizeof(int) );
	add<<<1,1>>>(2, 7, dev_c);

	// IMPORTANT:
	// Do not dereference pointer returned by cudaMalloc() from code that
	// executes on the host.
	//
	// Host code may pass this pointer around, perform arithmetic on it,
	// or even cast it to a different type, but you CAN NOT use it to read and
	// write from memory.

	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("2 + 7 = %d\n", c);
	hipFree(dev_c);

	return 0;
}

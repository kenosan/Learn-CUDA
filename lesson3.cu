#include <stdio.h>

int main(void) {
	hipDeviceProp_t prop;

	int count;
	hipGetDeviceCount(&count);
	printf("We have %d devices\n", count);

	for(int i = 0; i < count; i++) {
		hipGetDeviceProperties(&prop, i);

		printf("\t--- General information for device %i ---\n", i);
		printf("Name: %s\n", prop.name);
		printf("Compute capability: %d.%d\n", prop.major, prop.minor);
		printf("Clock rate: %d\n", prop.clockRate);

		printf("Device copy overlap: ");
		if(prop.deviceOverlap) {
			printf("Enabled\n");
		} else {
			printf("Disabled\n");
		}

		printf("Kernel execution timeout: ");
		if(prop.kernelExecTimeoutEnabled) {
			printf("Enabled\n");
		} else {
			printf("Disabled\n");
		}

		printf("\t--- Memory information for device %i ---\n", i);
		printf("Total global mem: %ld\n", prop.totalGlobalMem);
		printf("Total constant mem: %ld\n", prop.totalConstMem);
		printf("Max memory pitch: %ld\n", prop.memPitch);
		printf("Texture alignment: %ld\n", prop.textureAlignment);

		printf("\t--- MP information for device %i ---\n", i);
		printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}

	return 0;
}

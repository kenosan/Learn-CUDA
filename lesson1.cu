
#include <hip/hip_runtime.h>
#include <stdio.h>

////////////
//	Notes:
//
//	 host: CPU + system's memory
// device: GPU + system's memory


/**
 * A kernel is a function that executes on the device.
 *
 * __global__ alerts the compiler that a function should be compiled to run on
 * a device instead of the host.
 *
 */
__global__ void kernel(void) {

}


int main(void) {
	// <<<>>> denote arguments passed to the runtime system. They are not arguments
	// to the device code but are parameters that will influence how the runtime
	// will launch the device code
	kernel<<<1,1>>>();

	printf("Hi, I'm learning CUDA!\n");

	return 0;
}
